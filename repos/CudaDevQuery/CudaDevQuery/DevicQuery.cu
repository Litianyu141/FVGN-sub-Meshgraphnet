#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];
void printDeviceProp(const hipDeviceProp_t& prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

void GenerateNumbers(int* number, int size)    //��������0-9֮��������
{
	for (int i = 0; i < size; i++) {
		number[i] = rand() % 10;
	}
}

//CUDA ��ʼ��
bool InitCUDA()
{
	int count;

	//ȡ��֧��Cuda��װ�õ���Ŀ
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;

	for (i = 0; i < count; i++) {

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//��ӡ�豸��Ϣ
		printDeviceProp(prop);

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}


// __global__ ���� (GPU��ִ��) ����������
__global__ static void sumOfSquares(int* num, int* result, clock_t* time)
{
	int sum = 0;

	int i;

	clock_t start = clock();

	for (i = 0; i < DATA_SIZE; i++) {

		sum += num[i] * num[i] * num[i];

	}

	*result = sum;

	*time = clock() - start;

}





int main()
{

	//CUDA ��ʼ��
	if (!InitCUDA()) {
		return 0;
	}

	//���������
	GenerateNumbers(data, DATA_SIZE);

	/*�����ݸ��Ƶ��Կ��ڴ���*/
	int* gpudata, * result;

	clock_t* time;

	//hipMalloc ȡ��һ���Կ��ڴ� ( ����result�����洢��������time�����洢����ʱ�� )
	hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t));

	//hipMemcpy ����������������Ƶ��Կ��ڴ���
	//hipMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
	//hipMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	// ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
	sumOfSquares << <1, 1, 0 >> > (gpudata, result, time);


	/*�ѽ������ʾоƬ���ƻ����ڴ�*/

	int sum;
	clock_t time_used;

	//hipMemcpy ��������Դ��и��ƻ��ڴ�
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

	//Free
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	printf("GPUsum: %d time: %d\n", sum, time_used);

	sum = 0;

	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i] * data[i];
	}

	printf("CPUsum: %d \n", sum);

	system("pause");

	return 0;
}
